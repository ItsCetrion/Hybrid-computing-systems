#include "hip/hip_runtime.h"
#include <cuda_operations.h>
#include <hip/hip_runtime.h>


// Ядро вычислений
__global__ void addVectorsKernel(const float *d_a, const float *d_b, float *d_result, const int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        d_result[idx] = d_a[idx] + d_b[idx];
    }
}

void addVectorsExtended(const float *h_a, const float *h_b, float *h_result, const int n, const int blockSize, const int gridSize) {
    std::size_t array_size = n * sizeof(float);
    float *d_a, *d_b, *d_result;

    hipMalloc(&d_a, array_size);
    hipMalloc(&d_b, array_size);
    hipMalloc(&d_result, array_size);

    hipMemcpy(d_a, h_a, array_size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, array_size, hipMemcpyHostToDevice);

    addVectorsKernel<<<gridSize, blockSize>>>(d_a, d_b, d_result, n);

    hipMemcpy(h_result, d_result, array_size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);
}

void addVectorsOptimal(const float *h_a, const float *h_b, float *h_result, const int n) {
    int blockSize = 128;
    int gridSize = (n + blockSize - 1) / blockSize;
    addVectorsExtended(h_a, h_b, h_result, n, blockSize, gridSize);
}

void addVectorsKernelRun(const float *d_a, const float *d_b, float *d_result, const int n, const int blockSize, const int gridSize, float *kernel_ms) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    addVectorsKernel<<<gridSize, blockSize>>>(d_a, d_b, d_result, n);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(kernel_ms, start, stop);
}

void* cuda_malloc(std::size_t size) {
    void *ptr;
    hipMalloc(&ptr, size);
    return ptr;
}

void cuda_free(void *ptr) {
    hipFree(ptr);
}


void cuda_memcpy_host_to_device(void *dst, const void *src, std::size_t size) {
    hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
}


void cuda_memcpy_device_to_host(void *dst, const void *src, std::size_t size) {
    hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
}

